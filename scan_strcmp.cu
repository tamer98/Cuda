#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define BLOCK_DIM 1024 // number of threads in a block

/* Here we do an inclusive scan of 'array' in place.
   'size' is the number of elements in 'array'.
   it should be a power of 2.
 
   We assume that 'array' is in shared memory so that there is no need to 
   copy it to shared memory here.
    */
__device__ void scan_plus(int *array, int size)
{
   for (unsigned int stride=1; stride <= size/2; stride *= 2) {
        int v;
        if (threadIdx.x >= stride) {
            v = array[threadIdx.x - stride];
        }
        __syncthreads(); /* wait until all threads finish reading 
		                    an element */

        if (threadIdx.x >= stride)
            array[threadIdx.x] += v;

        __syncthreads(); /* wait until all threads finish updating an
		                    element */
     }
     
} // scan_plus

/*
   This kernel compares the two strings s1 and s2. Both strings are
   terminated with a null byte.
   The result is an integer:  0, if s1 and s2 are equal;
                              a negative value if s1 is less than s2;
                              a positive value if s1 is greater than s2
   The argument 'result' is used to "return" the result.
   The arguments n1, n2 indicate the number of characters in s1 and s2, respectively
    (including the null byte at the end).
             
   We assume that the number of threads in a block is >= max(n1,n2).  
*/
__global__ void my_strcmp(const char  *s1, int n1, const char *s2, int n2,  int *result)
{


 __shared__ int differentValues[BLOCK_DIM];
    int threadID = threadIdx.x;
    int stepSize = blockDim.x;
    int localDifference = 0;


	for (int index = threadID; index < min(n1, n2); index += stepSize) 
	{
   		 if (s1[index] != s2[index]) 
   		 {
   		 
       		   localDifference = s1[index] - s2[index];
       		   break;
       		   
    	}
    		
    }


  differentValues[threadID] = localDifference;
  __syncthreads();



  scan_plus(differentValues, BLOCK_DIM);
  __syncthreads();

  if (threadID == 0)
   {
   
     *result = differentValues[BLOCK_DIM - 1];
     
   }


}


int main(int argc, char **argv) 
{

	char *dev_s1, *dev_s2;
    int *dev_result;
#if 0
    char s1[] = "supercalifragilisticexpialidocious";
    char s2[] = "supercalifragilisticexpialidocious";
#endif
    const char *s1, *s2; 

    if (argc == 3) {
        s1 = strdup(argv[1]);
        s2 = strdup(argv[2]);
    }
    else if (argc == 1) {
        /* read 2 strings from the standard input */
        if (scanf("%ms %ms", &s1, &s2) != 2) {
            fprintf(stderr, "invalid input\n");
            exit(1);
        }
    }
    else {
        fprintf(stderr, "usage: %s [<first string> <second string>]\n", argv[0]);
        exit(1);
    }

    int n1 = strlen(s1)+1; // null byte at the end is also counted
    int n2 = strlen(s2)+1;
           
    // allocate the memory on the GPU
    hipMalloc((void**)&dev_s1, n1);
    hipMalloc((void**)&dev_s2, n2);
    hipMalloc((void**)&dev_result, sizeof(int));
    
    hipMemcpy(dev_s1, s1, n1, hipMemcpyHostToDevice);
    hipMemcpy(dev_s2, s2, n2, hipMemcpyHostToDevice);
    
    int threadsPerBlock = BLOCK_DIM;
    int numOfBlocks = 1;
 
    my_strcmp<<<numOfBlocks, threadsPerBlock>>>(dev_s1, n1, dev_s2, n2, dev_result);
 
    // copy the result back from the GPU to the CPU
    int result;
    hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("result is %d\n", result);
		
	    
    // free memory on the GPU side
    hipFree(dev_s1);
    hipFree(dev_s2);
    hipFree(dev_result);
}
